#include "hip/hip_runtime.h"
#include "Kernels.hpp"

__device__ std::size_t __get_idx()
{
    return (blockIdx.x * blockDim.x) + threadIdx.x;
}
__global__ void __copy_image(const pixel64_t* input, pixel64_t* output)
{
    const auto idx = __get_idx();
    output[idx] = input[idx];
}

__global__ void __sum(const pixel64_t* input, pixel64_t* output, std::size_t length)
{
    const auto idx = __get_idx();
    const auto offset = idx * length;
    pixel64_t sum = 0;
    for (std::size_t i = offset; i < offset + length; i++)
    {
        const auto p = input[i];
        sum += p;
    }

    output[idx] = sum;
}

__global__ void __pixel_minus_mean_pow2(const pixel64_t* input, pixel64_t* output, pixel64_t mean)
{
    const auto idx = __get_idx();

    const auto p = input[idx];
    const auto p_minus_mean = (long long) p - (long long) mean;
    output[idx] = p_minus_mean * p_minus_mean;
}

__global__ void __clear_image(pixel64_t* inout)
{
    const auto idx = __get_idx();

    const pixel64_t zero = 0;
    inout[idx] = zero;
}

__global__ void __threshold(const pixel64_t* input, pixel64_t* output, pixel64_t threshold)
{
    const auto idx = __get_idx();

    const auto t = threshold;
    const auto in = input[idx];

    if (in >= t)
        output[idx] = (pixel64_t) D_MAX_PIXEL_VAL;
}

__global__ void __sum_history(const pixel64_t* input, pixel64_t* output, std::size_t history_size, std::size_t area)
{
    const auto pixel_idx = __get_idx();

    pixel64_t sum = 0;
    for (std::size_t i = 0; i < history_size; i++)
    {
        const auto history_idx = i * area + pixel_idx;
        sum += input[history_idx];
    }
    output[pixel_idx] = sum;
}

__global__ void __remove_background(const pixel64_t* input, pixel64_t* output, pixel64_t div_val, pixel64_t tolerance)
{
    const auto idx = __get_idx();

    const int64_t mean = (int64_t) (input[idx] / div_val);
    const int64_t p = (int64_t) output[idx];
    if ((pixel64_t) std::abs(mean - p) < tolerance)
        output[idx] = 0;

}

__global__ void __lone_pixel(const pixel64_t* input, pixel64_t* output, std::size_t num_adjacent, std::size_t width)
{
    const auto idx = __get_idx();

    std::size_t idxs[8];
    idxs[0] = idx - 1;
    idxs[1] = idx + 1;
    idxs[2] = idx - width;
    idxs[3] = idx + width;
    idxs[4] = idxs[2] - 1;
    idxs[5] = idxs[2] + 1;
    idxs[6] = idxs[3] - 1;
    idxs[7] = idxs[3] + 1;
    pixel64_t sum = 0;
    for (std::size_t i = 0; i < 8; i++)
        sum += input[idxs[i]];

    if (sum < D_MAX_PIXEL_VAL * num_adjacent)
        output[idx] = 0;
}
